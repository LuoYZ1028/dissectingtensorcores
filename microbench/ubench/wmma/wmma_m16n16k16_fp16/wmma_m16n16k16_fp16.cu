#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include "../../../hw_def/hw_def.h"

// #define SHARED_MEM_SIZE (32 * 102 4) // 32 KB
// Launch only one thread to calcaulte the latency using a pointer-chasing
// array technique
//#define THREADS_NUM 32
// iterate over the array ITERS times

#ifndef ILPconfig
#define ILPconfig 1
#endif

#ifndef ITERS
#define ITERS  (1024)
#endif

using namespace nvcuda;
#define M 16
#define N 16
#define K 16
#define BLOCKS_NUM 1
#define WARP_SIZE 32
#define A_LAYOUT wmma::row_major
#define B_LAYOUT wmma::col_major
#define D_LAYOUT wmma::mem_row_major
// #define MAX_ACC 16

template <class T, class R>
__global__ void tensor161616_flops(uint64_t *startClk, uint64_t *stopClk, 
	  T *mat_a, T *mat_b, R *res, int M_GLOBAL, int N_GLOBAL, int K_GLOBAL, int warp_num) {
	int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;

	// declare fragments
	wmma::fragment<wmma::matrix_a, M, N, K, T, A_LAYOUT> a_frag;
	wmma::fragment<wmma::matrix_b, M, N, K, T, B_LAYOUT> b_frag;
	// wmma::fragment<wmma::accumulator, M, N, K, R> acc_frag[MAX_ACC];
	// for (int i = 0; i < MAX_ACC; i++)
	// 	wmma::fill_fragment(acc_frag[i], 0.0f);
	wmma::fragment<wmma::accumulator, M, N, K, R> acc_frag;
	wmma::fill_fragment(acc_frag, 0.0f);
	

	uint64_t start = 0;
	uint64_t stop = 0;

	asm volatile("bar.sync 0;");
	asm volatile("mov.u64 %0, %%clock64;" : "=l"(start)::"memory");
	// for (int ite = 0; ite < ITERS; ite++) {
	int a_row = warpId * M;
	// loop over n & k
	for (int n = 0; n < N_GLOBAL; n += N) {
		int b_col = n;
		for (int k = 0; k < K_GLOBAL; k += K) {
			int a_col = k;
			int b_row = k;
			// load input then perform MMA 
			if (a_row < M_GLOBAL && a_col < K_GLOBAL && b_row < K_GLOBAL && b_col < N_GLOBAL) {
				wmma::load_matrix_sync(a_frag, mat_a + a_row * K_GLOBAL + a_col, K_GLOBAL);
				wmma::load_matrix_sync(b_frag, mat_b + b_col * K_GLOBAL + b_row, K_GLOBAL);
				wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
			}
		}
		wmma::store_matrix_sync(res + a_row * N_GLOBAL + b_col, acc_frag, N_GLOBAL, D_LAYOUT);
	}
	// }
	__syncwarp();
	asm volatile("mov.u64 %0, %%clock64;" : "=l"(stop)::"memory");
	
	// int res_row = warpId * M;
	// int res_col = warpId * N;
	// if (res_row < M_GLOBAL && res_col < N_GLOBAL)
	// 	wmma::store_matrix_sync(res + res_row * N_GLOBAL + res_col, acc_frag, N_GLOBAL, D_LAYOUT);

	startClk[warpId] = start;
	stopClk[warpId] = stop;
}

// template <class T, class R> 
__host__ void mma_on_host(half *A, half *B, float *D, int M_GLOBAL, int N_GLOBAL, int K_GLOBAL) {
	for (int m = 0; m < M_GLOBAL; m++) {
		for (int n = 0; n < N_GLOBAL; n++) {
			float temp = 0.0f;
			for (int k = 0; k < K_GLOBAL; k++) {
				temp += __half2float(A[m * K_GLOBAL + k]) * __half2float(B[n * K_GLOBAL + k]);
			}
			D[m * N_GLOBAL + n] = temp;
		}
	}
}

template <class T, class R> 
float tensor161616_max_flops(int warp_num, bool report_fma_bw = false) {
	intilizeDeviceProp(0);
	int total_threads = warp_num * WARP_SIZE;

	uint32_t M_GLOBAL = M * warp_num;
	uint32_t N_GLOBAL = N;
	uint32_t K_GLOBAL = K;
	uint32_t A_GLOBAL = M_GLOBAL * K_GLOBAL;
	uint32_t B_GLOBAL = K_GLOBAL * N_GLOBAL;
	uint32_t D_GLOBAL = M_GLOBAL * N_GLOBAL;

	uint64_t *startClk = (uint64_t *)malloc(total_threads * sizeof(uint64_t));
	uint64_t *stopClk = (uint64_t *)malloc(total_threads * sizeof(uint64_t));
	T *data_a = (T *)malloc(A_GLOBAL * sizeof(T));
	T *data_b = (T *)malloc(B_GLOBAL * sizeof(T));
	R *cuda_res = (R *)malloc(D_GLOBAL * sizeof(R));
	R *cpu_res = (R *)malloc(D_GLOBAL * sizeof(R));

	uint64_t *startClk_ptr;
	uint64_t *stopClk_ptr;
	T *data_a_ptr;
	T *data_b_ptr;
	R *cuda_res_ptr;

	// 矩阵AB内，元素都是固定的数
	for (int i = 0; i < A_GLOBAL; i++) { data_a[i] = T(i / 16); }
	for (int i = 0; i < B_GLOBAL; i++) { data_b[i] = T(i / 16); }
	
	// 使用 hipMalloc 在 GPU 内分配空间，地址赋予 ptr
	gpuErrchk(hipMalloc(&startClk_ptr, total_threads * sizeof(uint64_t)));
	gpuErrchk(hipMalloc(&stopClk_ptr, total_threads * sizeof(uint64_t)));
	gpuErrchk(hipMalloc(&data_a_ptr, A_GLOBAL * sizeof(T)));
	gpuErrchk(hipMalloc(&data_b_ptr, B_GLOBAL * sizeof(T)));
	gpuErrchk(hipMalloc(&cuda_res_ptr, D_GLOBAL * sizeof(R)));
	// 将数据搬到上述 GPU 分配的空间
	gpuErrchk(hipMemcpy(data_a_ptr, data_a, 
		A_GLOBAL * sizeof(T), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(data_b_ptr, data_b, 
		B_GLOBAL * sizeof(T), hipMemcpyHostToDevice));
	// 给 mma 操作计时
	tensor161616_flops<T, R><<<BLOCKS_NUM, WARP_SIZE * warp_num>>>(
		startClk_ptr, stopClk_ptr, data_a_ptr, data_b_ptr, cuda_res_ptr,
		M_GLOBAL, N_GLOBAL, K_GLOBAL, warp_num);
	mma_on_host(data_a, data_b, cpu_res, M_GLOBAL, N_GLOBAL, K_GLOBAL);
	gpuErrchk(hipPeekAtLastError());
	// 没有发生错误才将 时间数据 和 乘法结果 放入 GPU 内部
	gpuErrchk(hipMemcpy(startClk, startClk_ptr, 
		total_threads * sizeof(uint64_t), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(stopClk, stopClk_ptr, 
		total_threads * sizeof(uint64_t), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(cuda_res, cuda_res_ptr, 
		D_GLOBAL * sizeof(R), hipMemcpyDeviceToHost));
  
	for (int i = 0 ; i < D_GLOBAL; i++) {
		if (fabs(cuda_res[i] - cpu_res[i]) > 0.01 * cpu_res[i])
       		printf("mismatch i=%d result_cuda=%f result_cpu=%f\n", i, cuda_res[i],
              cpu_res[i]);
	}

	// 总耗时 = 最晚结束时间 - 最早开始时间
	uint64_t total_time =
		*std::max_element(&stopClk[0], &stopClk[total_threads]) -
		*std::min_element(&startClk[0], &startClk[total_threads]);

	float fma_bw = ((float)(ITERS * M * N * K * ILPconfig * warp_num)) 
		/ (float)total_time;

	std::cout << "wmma-m" << M << "n" << N << "k" << K << \
		".row.row.fp16  latency " << (float)total_time/(float)ITERS << " cycles\n";
	std::cout << "FMA tensor bandwidth = " << fma_bw << "(FMA/clk/SM)\n";
	std::cout << "Total Clk number = " << total_time << std::endl;
	return 0.0;
}

int main() {
	std::vector<int> warps = {1, 2, 6, 8, 12, 16};
	intilizeDeviceProp(0);
	std::cout << "***********************************" << std::endl;
	std::cout << "wmma-m" << M << "n" << N << "k" << K << \
		".row.row.fp16 microbenchmark with ILP = " << ILPconfig << std::endl;

	for (auto& e:warps) {
		std::cout << "Number of warps = " << e << std::endl;
		tensor161616_max_flops<half, float>(e);
		std::cout << std::endl;
	}
	return 0;
}
  
