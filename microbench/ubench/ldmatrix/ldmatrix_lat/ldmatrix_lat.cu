#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "../../../hw_def/hw_def.h"

#define SHARED_MEM_SIZE (32 * 1024 / 4) // 32 KB
// Launch only one thread to calcaulte the latency using a pointer-chasing
// array technique
//#define THREADS_NUM 32
// iterate over the array ITERS times
#ifndef ITERS
#define ITERS  (1024 )
#endif


#ifndef ILPconfig
#define ILPconfig 1
#endif

static_assert(ILPconfig<=1,"ILP > 1 is not supported\n");

// two way bank conflict - > 23 latenct
// bank-conflict-free -> 25 latency

typedef uint32_t shared_m;
// Measure latency of ITERS ldmatrix.x1
__global__ void shared_lat(uint32_t *startClk, uint32_t *stopClk,
  shared_m *dsink, uint32_t stride) {

  // thread index
  uint32_t tid = threadIdx.x;
  uint32_t bid = blockIdx.x;
  uint32_t uid = bid * blockDim.x + tid;
  uint32_t n_threads = blockDim.x * gridDim.x;

  __shared__ shared_m s[SHARED_MEM_SIZE]; // static shared memory

  // one thread to initialize the pointer-chasing array
  if(uid == 0){
    for (uint32_t i = 0; i < (SHARED_MEM_SIZE - stride); i ++)
      s[i] = (i )*16 % 2048; // s[i] is multiple of 16, because addree is aligned with 4 bytes
  }
    
    asm volatile("bar.sync 0;");

    // if(uid == 0){
    //   for(int i = 0; i < SHARED_MEM_SIZE; i ++){
    //     printf("s[%d] = %d \t", i, s[i]);
  
    //   }
    //   printf("\n");
    // }
  //if (uid == 0) {
    // initalize pointer chaser
    //unsigned x = threadIdx.x*4;
    unsigned addr = static_cast<unsigned>(__cvta_generic_to_shared(&s[threadIdx.x*4]));
    //printf("thread %d , addr = %d \n", tid, addr);
    // start timing
    uint32_t start = 0;
    asm volatile("mov.u32 %0, %%clock;" : "=r"(start)::"memory");

    // pointer-chasing ITERS times
    //#pragma unroll
    for (uint32_t i = 0; i < ITERS; ++i) {
        asm volatile ("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];" : "=r"(addr) : "r"(addr)); // first 11
        __syncwarp();
    }
    //asm volatile("bar.sync 0;");
   
    //asm volatile ("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];" : "=r"(x) : "r"(addr));
    // stop timing
    uint32_t stop = 0;
    asm volatile("mov.u32 %0, %%clock;" : "=r"(stop)::"memory");
    addr ++;
    //printf("thread %d , x = %d \n", tid, addr);

    // write time and data back to memory
    if(uid == 0){

      startClk[uid] = start;
      stopClk[uid] = stop;
      dsink[uid] = addr;
    }


    // float lat = (float)(stopClk[0] - startClk[0]) / ITERS;
    // printf("Shared Memory Latency  = %f cycles\n", lat);
  //}
}


void test_with_different_thread(int THREADS_NUM){

  BLOCKS_NUM = 1;
  TOTAL_THREADS = THREADS_NUM * BLOCKS_NUM;
  THREADS_PER_SM = THREADS_NUM * BLOCKS_NUM;

  assert(SHARED_MEM_SIZE * sizeof(shared_m) < MAX_SHARED_MEM_SIZE_PER_BLOCK);

  uint32_t *startClk = (uint32_t *)malloc(sizeof(uint32_t));
  uint32_t *stopClk = (uint32_t *)malloc(sizeof(uint32_t));
  shared_m *dsink = (shared_m *)malloc(sizeof(shared_m));

  uint32_t *startClk_g;
  uint32_t *stopClk_g;
  shared_m *dsink_g;

  gpuErrchk(hipMalloc(&startClk_g, sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&stopClk_g, sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&dsink_g, sizeof(shared_m)));

  shared_lat<<<BLOCKS_NUM, THREADS_NUM>>>(startClk_g, stopClk_g, dsink_g, 1);
  gpuErrchk(hipPeekAtLastError());
    //printf("pass kenerl \n");
  gpuErrchk(hipMemcpy(startClk, startClk_g, sizeof(uint32_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(
      hipMemcpy(stopClk, stopClk_g, sizeof(uint32_t), hipMemcpyDeviceToHost));
  gpuErrchk(
      hipMemcpy(dsink, dsink_g, sizeof(shared_m), hipMemcpyDeviceToHost));

  float lat = (float)(stopClk[0] - startClk[0]) / ITERS;

  std::cout << THREADS_NUM/32 <<" warps ldmatrix.x1 latency " << lat <<" ( " <<(unsigned)(lat) << " ) " << std::endl;

  long num_bytes =  (THREADS_NUM/32) * 8 * 8 * 2 * 1;
  std::cout << "Shared mem throughput = " << num_bytes / lat << " bytes/clk " <<std::endl;
  std::cout << "Total Clk number " <<  stopClk[0] - startClk[0] <<std::endl;
  std::cout << std::endl;
  
  hipDeviceSynchronize();
 
  // printf("Shared Memory Latency  = %f cycles\n", lat);
  // printf("Total Clk number = %u \n", stopClk[0] - startClk[0]);

  // if (ACCEL_SIM_MODE) {
  //   std::cout << "\n//Accel_Sim config: \n";
  //   std::cout << "-gpgpu_smem_latency " << (unsigned)(lat) << std::endl;
  // }
  
}
int main() {
  intilizeDeviceProp(0);
  std::vector<int> warps = {1,2,4,8,16,32};
  std::cout << "ldmatrix.x1 microbenchmark " <<std::endl;
  for(auto &e:warps){
    test_with_different_thread(32*e);
  }

  return 0;
}
