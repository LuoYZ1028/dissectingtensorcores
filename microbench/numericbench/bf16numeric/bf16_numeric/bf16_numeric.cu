#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <random>
#include <cmath>
#include "../../../hw_def/hw_def.h"
#include "../../cpu_base.h"

typedef __hip_bfloat16 op_AB; 
typedef float op_CD; 




#ifndef MEAN
#define MEAN (0.0)
#endif

#ifndef STDDEV
#define STDDEV (1.0)
#endif

#ifndef N
#define N STDDEV //(1.0)
#endif



#ifndef ITERS
#define ITERS  (1024 )
#endif

#define ROUNDS  (ITERS*10 )

const int inst_m = 16;
const int inst_n = 8;
const int inst_k = 16;

__forceinline__ __device__ unsigned lane_id_()
{
    unsigned ret; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}

__global__ void gemm_m16n8k16_kernel(float* MatA,float* MatB,float* MatC, float* MatD ){

    constexpr const int inst_k = 16;

    uint32_t lane_id =  lane_id_();
    // four threads per group, group id
    uint32_t group_id = lane_id >>2;
    uint32_t tid_in_group = lane_id % 4;
    // m16 n8 k16
    __hip_bfloat16 frag_A[8]; // 16 * 16  / 32 = 8 * bf16
    __hip_bfloat16 frag_B[4]; // 8 * 16  / 32
    float frag_D[4]; // float , 16*8 /32 = 4*float
    // load operand fragA
    #pragma unroll
    for(int i =0; i < 8; i++){
        uint32_t row_a = 0;
        if( (i>=0 && i<2) || (i>=4 && i<6) ){
            row_a = group_id;
        }else{
            row_a = group_id + 8;
        }
        uint32_t col_a = 0;
        if(i<4){
            col_a = (tid_in_group * 2) + (i & 0x1);
        }else{
            col_a = (tid_in_group * 2) + (i & 0x1) + 8;
        }
        frag_A[i] = MatA[inst_k*row_a + col_a];
    }

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_b = 0;
        if( i < 2 ){
            row_b = (tid_in_group * 2) + (i & 0x1);
        }else{
            row_b = (tid_in_group * 2) + (i & 0x1)+8;
        }
        uint32_t col_b = group_id;
        frag_B[i] = MatB[row_b*inst_n + col_b];
    }

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_c = 0;
        if( i < 2 ){
            row_c = group_id;
        }else{
            row_c = group_id + 8;
        }
        uint32_t col_c = (tid_in_group * 2) + (i & 0x1);
        frag_D[i] = MatC[inst_n*row_c + col_c];
    }

    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
    float *C = reinterpret_cast<float *>(&frag_D[0]);
    float *D = C;  // D = A*B + D.
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 "
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), "r"(B[1]),
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
      );
    
    __syncwarp();
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_d = 0;
        if( i < 2 ){
            row_d = group_id;
        }else{
            row_d = group_id + 8;
        }
        uint32_t col_d = (tid_in_group * 2) + (i & 0x1);
        MatD[inst_n*row_d + col_d] = frag_D[i];
    }

}


__global__ void gemm_m16n8k8_kernel(float* MatA,float* MatB,float* MatC, float* MatD ){
    constexpr const int inst_k = 8;

    uint32_t lane_id =  lane_id_();
    uint32_t group_id = lane_id >>2;
    uint32_t tid_in_group = lane_id % 4;
    // m16 n8 k16
    __hip_bfloat16 frag_A[4]; // 16 * 16  / 32 = 8 * bf16
    __hip_bfloat16 frag_B[2]; // 8 * 16  / 32
    float frag_D[4]; // float , 16*8 /32 = 4*float
    // load operand fragA
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_a = 0;
        if( i==0 || i ==1 ){
            row_a = group_id;
        }else{
            row_a = group_id + 8;
        }

        uint32_t col_a = (tid_in_group * 2) + (i & 0x1);
        // row major
        frag_A[i] = MatA[inst_k*row_a + col_a];
        
    }
    // load operand fragB, MatB has to be col-major
    #pragma unroll
    for(int i =0; i < 2; i++){
        uint32_t row_b =  (tid_in_group * 2) + (i);

        uint32_t col_b = group_id;
        // row-major B
        frag_B[i] = MatB[row_b*inst_n + col_b];
    }

    // load operand fragC, MatC has to be row-major
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_c = 0;
        if( i < 2 ){
            row_c = group_id;
        }else{
            row_c = group_id + 8;
        }
        uint32_t col_c = (tid_in_group * 2) + (i & 0x1);
        // row-major
        frag_D[i] = MatC[inst_n*row_c + col_c];
    }

    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
    float *C = reinterpret_cast<float *>(&frag_D[0]);
    float *D = C;  // D = A*B + D.

    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.bf16.bf16.f32 "
        "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), // "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), //"r"(B[1]),
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
      );
    
    __syncwarp();

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_d = 0;
        if( i < 2 ){
            row_d = group_id;
        }else{
            row_d = group_id + 8;
        }
        uint32_t col_d = (tid_in_group * 2) + (i & 0x1);
        // row-major
        MatD[inst_n*row_d + col_d] = frag_D[i];
    }

}




std::vector<double> numeric_bf16_bench(){


    int BLOCKS_NUM = 1;
    int nwarps = 1;
    int warp_size = 32;

    
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;


    float *host_matA = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC = (float *)malloc(total_C_SIZE * sizeof(float));
    float *host_matD = (float *)malloc(total_C_SIZE * sizeof(float));

    std::random_device rd{};
    std::mt19937 gen{rd()};
    std::uniform_real_distribution<float> random_gen{ (float) std::pow(2,float(N-1) ) ,(float)std::pow(2,float(N))};
    //std::uniform_real_distribution<float> random_gen{ 0,1};
    // initialize A, row-major
    float *host_matA_cpu = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB_cpu = (float *)malloc(total_B_SIZE * sizeof(float));
    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_k; c ++){
            //float rnd = (float)(r*inst_k+c);
            float rnd = (float)random_gen(gen);
            host_matA_cpu[r*inst_k+c] = rnd;
            host_matA[r*inst_k+c] = (op_AB)rnd;
        }
    }
    // std::cout<<"print MatA" <<std::endl;
    // print_mat(host_matA_cpu,inst_m,inst_k);
    // initialize B, row-major
    for(int r = 0; r < inst_k; r ++){
        for(int c = 0; c < inst_n; c ++){
            float rnd = (float)random_gen(gen);
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            host_matB_cpu[r*inst_n+c] = rnd;
            host_matB[r*inst_n+c] = (op_AB)rnd;
        }
    }
    // initialize C, row-major
    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_n; c ++){
            host_matC[r*inst_n+c] = 0.0 ;//  (op_CD)random_gen(gen);
        }
    }



    float *cpu_res_baseline = (float *)malloc(total_C_SIZE * sizeof(float));
    // host computation
    gemm_mnk_cpu(host_matA_cpu,host_matB_cpu,host_matC,cpu_res_baseline,inst_m,inst_n,inst_k);


    /***************************************** m16n8k8 *******************************************/
    float *dev_matA;
    float *dev_matB;
    float *dev_matC;

    float *dev_matD;
    // allocate device global memory
    // D = A*B + C
    hipMalloc(&dev_matA, total_A_SIZE * sizeof(float));
    hipMalloc(&dev_matB, total_B_SIZE * sizeof(float));
    hipMalloc(&dev_matC, total_C_SIZE * sizeof(float));
    hipMalloc(&dev_matD, total_C_SIZE * sizeof(float));
    // copy data from host to device
    gpuErrchk(hipMemcpy(dev_matA, host_matA, total_A_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(dev_matB, host_matB, total_B_SIZE * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_matC, host_matC, total_C_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gemm_m16n8k16_kernel<<<BLOCKS_NUM, nwarps*warp_size>>>(dev_matA,dev_matB,dev_matC,dev_matD);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(host_matD, dev_matD, total_C_SIZE * sizeof(float), hipMemcpyDeviceToHost));

    /***************************************** m16n8k4 *******************************************/


    float *host_matA_halfK = (float *)malloc(total_A_SIZE * sizeof(float)/2 );
    float *host_matB_halfK = (float *)malloc(total_B_SIZE * sizeof(float)/2);

    // store temp result
    float *host_matD_halfK = (float *)malloc(total_C_SIZE * sizeof(float));

    // final result
    float *host_matD_final_halfK = (float *)malloc(total_C_SIZE * sizeof(float));

    for(int i = 0; i < inst_m*inst_n; i ++){
        host_matD_final_halfK[i] = 0.0f;
        host_matD_halfK[i] = 0.0f;
    }

    //std::memset(host_matD_final_halfK, 0.0, total_C_SIZE * sizeof(op_CD));
    float *dev_matA_halfK;
    float *dev_matB_halfK;
    float *dev_matC_halfK;

    float *dev_matD_halfK;

    hipMalloc(&dev_matA_halfK, total_A_SIZE * sizeof(float)/2 );
    hipMalloc(&dev_matB_halfK, total_B_SIZE * sizeof(float)/2 );
    hipMalloc(&dev_matC_halfK, total_C_SIZE * sizeof(float));
    hipMalloc(&dev_matD_halfK, total_C_SIZE * sizeof(float));




    for(int i =0; i <2; i++){
        // copy half K to
        // mat a 
        for(int row = 0; row < inst_m; row ++){
            for(int col = 0; col < inst_k/2; col ++){
                host_matA_halfK[col + row*inst_k/2] = host_matA_cpu[col + row*inst_k + (inst_k/2) * i];
            }
        }

        for(int row = 0; row < inst_k/2; row ++){
            for(int col = 0; col < inst_n; col ++){
                host_matB_halfK[col + row*inst_n] = host_matB_cpu[col + (row + i * inst_k/2)*inst_n ];
            }
        }



        // copy data from host to device
        gpuErrchk(hipMemcpy(dev_matA_halfK, host_matA_halfK, total_A_SIZE * sizeof(float) / 2, hipMemcpyHostToDevice));
    
        gpuErrchk(hipMemcpy(dev_matB_halfK, host_matB_halfK, total_B_SIZE * sizeof(float) / 2, hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(dev_matC_halfK, host_matC, total_C_SIZE * sizeof(float), hipMemcpyHostToDevice));

        gemm_m16n8k8_kernel<<<BLOCKS_NUM, nwarps*warp_size>>>(dev_matA_halfK,dev_matB_halfK,dev_matC_halfK,dev_matD_halfK);
        gpuErrchk(hipPeekAtLastError());
    
        gpuErrchk(hipMemcpy(host_matD_halfK, dev_matD_halfK, total_C_SIZE * sizeof(float), hipMemcpyDeviceToHost));
        
        hipDeviceSynchronize();
        for(int n = 0; n < inst_m*inst_n; n ++){
            host_matD_final_halfK[n] += host_matD_halfK[n];
        }

    }


    double l2_relative_err = 0.0;
    compute_diff_l2_norm(cpu_res_baseline,host_matD,inst_m,inst_n,l2_relative_err);


    double l2_relative_err_halfK = 0.0;
    compute_diff_l2_norm(cpu_res_baseline,host_matD_final_halfK,inst_m,inst_n,l2_relative_err_halfK);

    std::vector<double> res{l2_relative_err,l2_relative_err_halfK};

    return res;

}

int main(){
    std::cout<<"***********************************"<<std::endl;
    std::cout << "mma bf16 numeric errors w.r.t fp32 on cpu "  << std::endl;
    //std::cout << "Random initialization with normal_distribution, mean = " << MEAN << ", stddev = " <<STDDEV  << std::endl;
    std::cout << "Random initialization range [ " << std::pow(2,float(N-1) )<<","<<std::pow(2,float(N) )<<"]"  << std::endl;

    double l2_relative_err = 0.0;
    double l2_relative_err_halfK = 0.0;
    
    for(int i=0;i < ROUNDS; i ++){
        std::vector<double> errors = numeric_bf16_bench();
        l2_relative_err += errors[0];
        l2_relative_err_halfK += errors[1];
    }

    // std::cout<<"element-wise error :"<<std::endl;
    // std::cout<<"element-wise avg_abs_err = " << avg_abs_err/ITERS <<std::endl;
    // std::cout<<"element-wise avg_l1_norm_err = " << avg_l1_norm/ITERS <<std::endl;

    std::cout<<"mma.m16n8k16.bf16 l2 relative error :"<< l2_relative_err/ROUNDS << std::endl;
    std::cout<<"mma.m16n8k8.bf16 l2 relative error :"<< l2_relative_err_halfK/ROUNDS << std::endl;

    std::cout<<"mma.m16n8k16.bf16 l2 relative error per FMA :"<< l2_relative_err/(ROUNDS*inst_m*inst_k*inst_n) << std::endl;
    std::cout<<"mma.m16n8k8.bf16 l2 relative error per FMA :"<< l2_relative_err_halfK/(ROUNDS*inst_m*inst_k*inst_n) << std::endl;

}