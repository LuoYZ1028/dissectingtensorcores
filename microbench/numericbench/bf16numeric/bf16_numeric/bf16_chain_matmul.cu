#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <random>
#include <cmath>
#include "../../../hw_def/hw_def.h"
#include "../../cpu_base.h"

typedef __hip_bfloat16 init_type;
#define Init_With_BF16 1

// typedef float init_type;
// #define Init_With_BF16 0


// #ifndef ITERS
// #define ITERS  (1024 )
// #endif

#define ROUNDS  (1000 )

#ifndef CHAINs
#define CHAINs  (50)
#endif


#ifndef MEAN
#define MEAN (0.0)
#endif

#ifndef STDDEV
#define STDDEV (1.0)
#endif


#define SEED 123456

struct random_generator{
    // int seed_;
    // float mean_;
    // float stddev_;
    std::mt19937 gen;
    std::normal_distribution<float> random_gen{MEAN,STDDEV};
    //std::uniform_real_distribution<> random_gen{-1.0,1.0};
    random_generator(int seed_){
        gen.seed(seed_);
    }

    float operator()(){
        return random_gen(gen);
    };
};




const int inst_m = 16;
const int inst_n = 8;
// const int inst_k = 8;

__forceinline__ __device__ unsigned lane_id_()
{
    unsigned ret; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}

__global__ void gemm_m16n8k16_kernel(float* MatA,float* MatB,float* MatC, float* MatD ){

    constexpr const int inst_k = 16;

    uint32_t lane_id =  lane_id_();
    // four threads per group, group id
    uint32_t group_id = lane_id >>2;
    uint32_t tid_in_group = lane_id % 4;
    // m16 n8 k16
    __hip_bfloat16 frag_A[8]; // 16 * 16  / 32 = 8 * bf16
    __hip_bfloat16 frag_B[4]; // 8 * 16  / 32
    float frag_D[4]; // float , 16*8 /32 = 4*float
    // load operand fragA
    #pragma unroll
    for(int i =0; i < 8; i++){
        uint32_t row_a = 0;
        if( (i>=0 && i<2) || (i>=4 && i<6) ){
            row_a = group_id;
        }else{
            row_a = group_id + 8;
        }
        uint32_t col_a = 0;
        if(i<4){
            col_a = (tid_in_group * 2) + (i & 0x1);
        }else{
            col_a = (tid_in_group * 2) + (i & 0x1) + 8;
        }
        frag_A[i] = MatA[inst_k*row_a + col_a];
    }

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_b = 0;
        if( i < 2 ){
            row_b = (tid_in_group * 2) + (i & 0x1);
        }else{
            row_b = (tid_in_group * 2) + (i & 0x1)+8;
        }
        uint32_t col_b = group_id;
        frag_B[i] = MatB[row_b*inst_n + col_b];
    }

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_c = 0;
        if( i < 2 ){
            row_c = group_id;
        }else{
            row_c = group_id + 8;
        }
        uint32_t col_c = (tid_in_group * 2) + (i & 0x1);
        frag_D[i] = MatC[inst_n*row_c + col_c];
    }

    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
    float *C = reinterpret_cast<float *>(&frag_D[0]);
    float *D = C;  // D = A*B + D.
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 "
        "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), "r"(B[1]),
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
      );
    
    __syncwarp();
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_d = 0;
        if( i < 2 ){
            row_d = group_id;
        }else{
            row_d = group_id + 8;
        }
        uint32_t col_d = (tid_in_group * 2) + (i & 0x1);
        MatD[inst_n*row_d + col_d] = frag_D[i];
    }

}


__global__ void gemm_m16n8k8_kernel(float* MatA,float* MatB,float* MatC, float* MatD ){
    constexpr const int inst_k = 8;

    uint32_t lane_id =  lane_id_();
    uint32_t group_id = lane_id >>2;
    uint32_t tid_in_group = lane_id % 4;
    // m16 n8 k16
    __hip_bfloat16 frag_A[4]; // 16 * 16  / 32 = 8 * bf16
    __hip_bfloat16 frag_B[2]; // 8 * 16  / 32
    float frag_D[4]; // float , 16*8 /32 = 4*float
    // load operand fragA
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_a = 0;
        if( i==0 || i ==1 ){
            row_a = group_id;
        }else{
            row_a = group_id + 8;
        }

        uint32_t col_a = (tid_in_group * 2) + (i & 0x1);
        // row major
        frag_A[i] = MatA[inst_k*row_a + col_a];
        
    }
    // load operand fragB, MatB has to be col-major
    #pragma unroll
    for(int i =0; i < 2; i++){
        uint32_t row_b =  (tid_in_group * 2) + (i);

        uint32_t col_b = group_id;
        // row-major B
        frag_B[i] = MatB[row_b*inst_n + col_b];
    }

    // load operand fragC, MatC has to be row-major
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_c = 0;
        if( i < 2 ){
            row_c = group_id;
        }else{
            row_c = group_id + 8;
        }
        uint32_t col_c = (tid_in_group * 2) + (i & 0x1);
        // row-major
        frag_D[i] = MatC[inst_n*row_c + col_c];
    }

    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
    float *C = reinterpret_cast<float *>(&frag_D[0]);
    float *D = C;  // D = A*B + D.

    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.bf16.bf16.f32 "
        "{%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), // "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), //"r"(B[1]),
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
      );
    
    __syncwarp();

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_d = 0;
        if( i < 2 ){
            row_d = group_id;
        }else{
            row_d = group_id + 8;
        }
        uint32_t col_d = (tid_in_group * 2) + (i & 0x1);
        // row-major
        MatD[inst_n*row_d + col_d] = frag_D[i];
    }

}



void gpu_bf16_m16n8k8(float* matA_in, float * matB_in, float* matC_in, float* matD_out){

    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;

    int BLOCKS_NUM = 1;
    int nwarps = 1;
    int warp_size = 32;
    
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    //*********** m16n8k8 device mem allocation***//
    // we use float to accloacte mem, since we will do the data conversion in the GPU side.
    // Note conversion in GPU side is not efficient, but it is fine for numeric profiling
    float *dev_matA;
    float *dev_matB;
    float *dev_matC;

    float *dev_matD;
    // allocate device global memory
    // D = A*B + C
    hipMalloc(&dev_matA, total_A_SIZE * sizeof(float));
    hipMalloc(&dev_matB, total_B_SIZE * sizeof(float));
    hipMalloc(&dev_matC, total_C_SIZE * sizeof(float));
    hipMalloc(&dev_matD, total_C_SIZE * sizeof(float));

    /***************************************** m16n8k8 *******************************************/
    // copy data from host to device
    gpuErrchk(hipMemcpy(dev_matA, matA_in, total_A_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(dev_matB, matB_in, total_B_SIZE * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_matC, matC_in, total_C_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gemm_m16n8k8_kernel<<<BLOCKS_NUM, nwarps*warp_size>>>(dev_matA,dev_matB,dev_matC,dev_matD);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(matD_out, dev_matD, total_C_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    hipFree(dev_matA);
    hipFree(dev_matB);
    hipFree(dev_matC);
    hipFree(dev_matD);

}


template<typename typeDest,typename typeSrc >
void copy_mat(typeDest* dest, typeSrc* source, int num_eles){
    for(int i=0;i<num_eles;i++){
        dest[i] = typeDest(source[i]);
    }
}


std::vector<double> bf16_chain_matmul(int n,random_generator & random_gen){

    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;
    // int BLOCKS_NUM = 1;
    int nwarps = 1;
    //int warp_size = 32;
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    int num_chain = n;

    float *host_matA = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC = (float *)malloc(total_C_SIZE * sizeof(float));
    float *host_matD = (float *)malloc(total_C_SIZE * sizeof(float));

    // initialize A, row-major
    float *host_matA_cpu = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB_cpu = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC_cpu = (float *)malloc(total_C_SIZE * sizeof(float));

    // initialize MatA
    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_k; c ++){
            //float rnd = (float)(r*inst_k+c);
            float rnd = (init_type )random_gen();
            host_matA_cpu[r*inst_k+c] = (float)rnd;
            host_matA[r*inst_k+c] = (float)rnd;
        }
    }
    // initialize MatB
    for(int r = 0; r < inst_k; r ++){
        for(int c = 0; c < inst_n; c ++){
            float rnd = (init_type )random_gen();
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            host_matB_cpu[r*inst_n+c] = (float)rnd;
            host_matB[r*inst_n+c] = (float)rnd;
        }
    }
    // initialize MatC
    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_n; c ++){
            float rnd = 0.0;  //(float)random_gen(gen);
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            //host_matB_cpu[r*inst_n+c] = rnd;
            host_matC_cpu[r*inst_n+c] = (float)rnd;
            host_matC[r*inst_n+c] = float(rnd); //0.0 ;//  (op_CD)random_gen(gen);
        }
    }

    float *cpu_res_baseline = (float *)malloc(total_C_SIZE * sizeof(float));
    float *gpu_m16n8k8 = (float *)malloc(total_C_SIZE * sizeof(float));
    //float *gpu_m16n8k4 = (op_CD *)malloc(total_C_SIZE * sizeof(op_CD));
    // host computation

    for(int chain_id =0; chain_id < num_chain; chain_id ++){

        gemm_mnk_cpu(host_matA_cpu,host_matB_cpu,host_matC_cpu,cpu_res_baseline,inst_m,inst_n,inst_k);
        //gpu 
        // gpu tf32 m16b8k8
        gpu_bf16_m16n8k8(host_matA,host_matB,host_matC,gpu_m16n8k8); //(float* matA_in, float * matB_in, float* matC_in, float* matD_out)
        // gpu tf32 m16b8k4
        //gpu_tf32_m16n8k4(host_matA,host_matB,host_matC,gpu_m16n8k4); // (float* matA_in, float * matB_in, float* matC_in, float* matD_out)
        // destination, source
        //copy_mat(host_matA,gpu_m16n8k4); 
        copy_mat(host_matA,gpu_m16n8k8,inst_m*inst_n);
        copy_mat(host_matA_cpu,cpu_res_baseline,inst_m*inst_n);
        // generate new matB and matC

        if(Mat_has_inf(cpu_res_baseline,inst_m*inst_n)){
            
            std::cout<<"CPU baseline matrix contains inf, break at chain = "<< chain_id+1 << " - " <<num_chain+1 <<std::endl;
            exit(0);
        }
        if(Mat_has_inf(gpu_m16n8k8,inst_m*inst_n)){
            
            std::cout<<"GPU matrix contains inf, break at chain = "<< chain_id+1 << " - " <<num_chain+1 <<std::endl;
            exit(0);
        }

        for(int r = 0; r < inst_k; r ++){
            for(int c = 0; c < inst_n; c ++){
                float rnd = (init_type)random_gen();
                //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
                host_matB_cpu[r*inst_n+c] =(float)rnd;
                host_matB[r*inst_n+c] = (float)rnd;
            }
        }
        // initialize C, row-major
        for(int r = 0; r < inst_m; r ++){
            for(int c = 0; c < inst_n; c ++){
                float rnd = 0.0;  //(float)random_gen(gen);
                //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
                //host_matB_cpu[r*inst_n+c] = rnd;
                host_matC_cpu[r*inst_n+c] = (float)rnd;
                host_matC[r*inst_n+c] = float(rnd); //0.0 ;//  (op_CD)random_gen(gen);
            }
        }

    }

    double l2_relative_err = 0.0;
    compute_diff_l2_norm(cpu_res_baseline,gpu_m16n8k8,inst_m,inst_n,l2_relative_err);
    // double l2_relative_err_halfK = 0.0;
    // compute_diff_l2_norm(cpu_res_baseline,host_matD_final_halfK,inst_m,inst_n,l2_relative_err_halfK);
    std::vector<double> res{l2_relative_err};
    return res;
}



int main(){
    // const int inst_m = 16;
    // const int inst_n = 8;
    // const int inst_k = 8;
    std::cout<<"***********************************"<<std::endl;
    std::cout << "mma bf16 numeric errors with chain matmul"  << std::endl;
    if(Init_With_BF16 == 1){
        std::cout<<"Initialization with bf16"<<std::endl;
    }else{
        std::cout<<"Initialization with fp32"<<std::endl;
    }

    
    for(int num_chain =1; num_chain<=CHAINs; num_chain++){
        // std::cout<<"***********************************"<<std::endl;
        // std::cout << "mma bf16 numeric errors chain matmul with number of chains = "  <<num_chain << std::endl;

        double l2_relative_err = 0.0;
        random_generator random_gen(SEED);
        for(int i=0;i < ROUNDS; i ++){
            std::vector<double> errors = bf16_chain_matmul(num_chain,random_gen);
            l2_relative_err += errors[0];
            // l2_relative_err_halfK += errors[1];
        }
        std::cout<<"length of chain = "<< num_chain <<", mma.m16n8k8.bf16 l2 relative error :"<< l2_relative_err/ROUNDS << std::endl;
        //std::cout<<"mma.m16n8k8.bf16 l2 relative error per FMA :"<< l2_relative_err/(ROUNDS*inst_m*inst_k*inst_n) << std::endl;
    }
}


/************ chain matmul pseudocode  **************/
// // Define initialization data type
// typedef init_type bf16; // bf16 or fp32
// // Define the chain length
// const int n_chains = N; // e.g. N= 50
// // Initialization for A and B
// initialize_random<init_type>(A_tmp);
// initialize_random<init_type>(B_tmp);
// // Copy to CPU
// Copy_CPU<fp32>(A_CPU,A_tmp);
// Copy_CPU<fp32>(B_CPU,B_tmp);
// // Copy to GPU
// Copy_GPU<bf16>(A_TC,A_tmp);
// Copy_GPU<bf16>(B_TC,B_tmp);
// // Start chain matmul
// for(int i=0; i<n_chains;i++){
//     D_CPU_fp32 = A_CPU*B_CPU ; // run on CPU
//     D_TC = A_TC*B_TC; // run on Tensor Cores
//     // Assign the current result to next Matrix A
//     Copy_CPU<fp32>(A_CPU,D_CPU_fp32);
//     Copy_GPU<bf16>(A_TC,D_TC);
//     // Generate new Matrix B for next round computation
//     initialize_random<init_type>(B_tmp);
//     Copy_CPU<fp32>(B_CPU,B_tmp);
//     Copy_GPU<bf16>(B_TC,B_tmp);
// }
// // compute the l2_relative error
// l2_err_TC_CPUfp32 = l2_relative_err(D_CPU_fp32,D_TC);

/************ chain matmul pseudocode **************/