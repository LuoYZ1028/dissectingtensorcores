#include "hip/hip_runtime.h"
// one m16n8k8_tf32 = two m16n8k4 
// see the errors for a same m16n8k8
// chain matrix multiplication
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <random>
#include "../../../hw_def/hw_def.h"
#include "../../cpu_base.h"
#include <cstring>


// typedef float op_AB; 
// typedef float op_CD; 


// typedef float init_type;
// #define Init_With_TF32 1

typedef float init_type;
#define Init_With_TF32 0



// #ifndef ITERS
// #define ITERS  (1024 )
// #endif

#define ROUNDS  (1000 )

#ifndef CHAINs
#define CHAINs  (50)
#endif


#ifndef MEAN
#define MEAN (0.0)
#endif

#ifndef STDDEV
#define STDDEV (1.0)
#endif


#define SEED 123456

struct random_generator{
    // int seed_;
    // float mean_;
    // float stddev_;
    std::mt19937 gen;
    std::normal_distribution<float> random_gen{MEAN,STDDEV};
    //std::uniform_real_distribution<> random_gen{-1.0,1.0};
    random_generator(int seed_){
        gen.seed(seed_);
    }

    float operator()(){
        return random_gen(gen);
    };
};




const int inst_m = 16;
const int inst_n = 8;
const int inst_k = 8;

__forceinline__ __device__ unsigned lane_id_()
{
    unsigned ret; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}


__global__ void gemm_m16n8k4_kernel(float* MatA,float* MatB,float* MatC, float* MatD ){

    constexpr const int inst_k = 4;
    uint32_t lane_id =  lane_id_();
    // four threads per group, group id
    uint32_t group_id = lane_id >>2;
    uint32_t tid_in_group = lane_id % 4;
    // m16 n8 k4
    uint32_t frag_A[2]; // 16 * 16  / 32 = 8 * bf16
    uint32_t frag_B[1]; // 8 * 16  / 32
    float frag_D[4]; // float , 16*8 /32 = 4*float
    // load operand fragA
    #pragma unroll
    for(int i =0; i < 2; i++){
        uint32_t row_a = 0;
        uint32_t col_a = 0;
        if( i==0 ){
            row_a = group_id;
        }else{
            row_a = group_id + 8;
        }
        col_a = tid_in_group;
        // row major
        // Cvt Float - TF32
        asm("cvt.rna.tf32.f32  %0, %1;\n" : "=r"(frag_A[i]) : "f"(MatA[inst_k*row_a + col_a]));
    }
    #pragma unroll
    for(int i =0; i < 1; i++){
        uint32_t row_b =  tid_in_group ; 
        uint32_t col_b = group_id;
        // row-major B
        asm("cvt.rna.tf32.f32  %0, %1;\n" : "=r"(frag_B[i]) : "f"(MatB[row_b*inst_n + col_b]));
        //frag_B[i] = (MatB[row_b*inst_n + col_b]);
    }

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_c = 0;
        if( i < 2 ){
            row_c = group_id;
        }else{
            row_c = group_id + 8;
        }
        uint32_t col_c = (tid_in_group * 2) + (i & 0x1);
        // row-major
        frag_D[i] = MatC[inst_n*row_c + col_c];
    }

    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
    float *C = reinterpret_cast<float *>(&frag_D[0]);
    float *D = C;  // D = A*B + D.

    asm volatile(
        "mma.sync.aligned.m16n8k4.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5}, {%6}, {%7,%8,%9,%10};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), 
          "r"(B[0]), 
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
    );
    
    __syncwarp();

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_d = 0;
        if( i < 2 ){
            row_d = group_id;
        }else{
            row_d = group_id + 8;
        }
        uint32_t col_d = (tid_in_group * 2) + (i & 0x1);
        // row-major
        MatD[inst_n*row_d + col_d] = frag_D[i];
    }
}



__global__ void gemm_m16n8k8_kernel(float* MatA,float* MatB,float* MatC, float* MatD ){
    // constexpr const int inst_m = 16;
    // constexpr const int inst_n = 8;
    constexpr const int inst_k = 8;
    uint32_t lane_id =  lane_id_();
    // four threads per group, group id
    uint32_t group_id = lane_id >>2;
    uint32_t tid_in_group = lane_id % 4;
    // m16 n8 k8
    uint32_t frag_A[4]; // 16 * 16  / 32 = 8 * bf16
    uint32_t frag_B[2]; // 8 * 16  / 32
    float frag_D[4]; // float , 16*8 /32 = 4*float
    // load operand fragA
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_a = 0;
        uint32_t col_a = 0;
        if( i==0 || i ==2 ){
            row_a = group_id;
        }else{
            row_a = group_id + 8;
        }
        if(i == 0 || i==1){// i ==0 || i ==2
            col_a = tid_in_group;
        }else{
            col_a = tid_in_group + 4;
        }
        asm("cvt.rna.tf32.f32  %0, %1;\n" : "=r"(frag_A[i]) : "f"(MatA[inst_k*row_a + col_a]));
    }

    #pragma unroll
    for(int i =0; i < 2; i++){
        uint32_t row_b =  0 ; //(i==0)?tid_in_group:(tid_in_group+4);//  (tid_in_group * 2) + (i);
        if(i == 0){
            row_b = tid_in_group;
        }else{
            row_b = tid_in_group + 4;
        }
        uint32_t col_b = group_id;
        // row-major B
        asm("cvt.rna.tf32.f32  %0, %1;\n" : "=r"(frag_B[i]) : "f"(MatB[row_b*inst_n + col_b]));
        //frag_B[i] = (MatB[row_b*inst_n + col_b]);
    }

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_c = 0;
        if( i < 2 ){
            row_c = group_id;
        }else{
            row_c = group_id + 8;
        }
        uint32_t col_c = (tid_in_group * 2) + (i & 0x1);
        // row-major
        frag_D[i] = MatC[inst_n*row_c + col_c];
    }

    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
    float *C = reinterpret_cast<float *>(&frag_D[0]);
    float *D = C;  // D = A*B + D.

    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5, %6, %7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), "r"(B[1]), 
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
    );
    
    __syncwarp();

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_d = 0;
        if( i < 2 ){
            row_d = group_id;
        }else{
            row_d = group_id + 8;
        }
        uint32_t col_d = (tid_in_group * 2) + (i & 0x1);
        // row-major
        MatD[inst_n*row_d + col_d] = frag_D[i];
    }

}

template<typename typeDest,typename typeSrc >
void copy_mat(typeDest* dest, typeSrc* source, int num_eles){
    for(int i=0;i<num_eles;i++){
        dest[i] = typeDest(source[i]);
    }
}



void gpu_tf32_m16n8k8(float* matA_in, float * matB_in, float* matC_in, float* matD_out){
    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;

    int BLOCKS_NUM = 1;
    int nwarps = 1;
    int warp_size = 32;
    
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    //*********** m16n8k8 device mem allocation***//
    float *dev_matA;
    float *dev_matB;
    float *dev_matC;

    float *dev_matD;
    // allocate device global memory
    // D = A*B + C
    hipMalloc(&dev_matA, total_A_SIZE * sizeof(float));
    hipMalloc(&dev_matB, total_B_SIZE * sizeof(float));
    hipMalloc(&dev_matC, total_C_SIZE * sizeof(float));
    hipMalloc(&dev_matD, total_C_SIZE * sizeof(float));

    /***************************************** m16n8k8 *******************************************/
    // copy data from host to device
    gpuErrchk(hipMemcpy(dev_matA, matA_in, total_A_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(dev_matB, matB_in, total_B_SIZE * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_matC, matC_in, total_C_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gemm_m16n8k8_kernel<<<BLOCKS_NUM, nwarps*warp_size>>>(dev_matA,dev_matB,dev_matC,dev_matD);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(matD_out, dev_matD, total_C_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    hipFree(dev_matA);
    hipFree(dev_matB);
    hipFree(dev_matC);
    hipFree(dev_matD);
}




void gpu_tf32_m16n8k4(float* matA_in, float * matB_in, float* matC_in, float* matD_out){
    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;

    int BLOCKS_NUM = 1;
    int nwarps = 1;
    int warp_size = 32;
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    //*********** m16n8k4 device mem allocation***//
    float *dev_matA_halfK;
    float *dev_matB_halfK;
    float *dev_matC_halfK;

    float *dev_matD_halfK;

    hipMalloc(&dev_matA_halfK, total_A_SIZE * sizeof(float)/2 );
    hipMalloc(&dev_matB_halfK, total_B_SIZE * sizeof(float)/2 );
    hipMalloc(&dev_matC_halfK, total_C_SIZE * sizeof(float));
    hipMalloc(&dev_matD_halfK, total_C_SIZE * sizeof(float));

    float *host_matA_halfK = (float *)malloc(total_A_SIZE * sizeof(float)/2 );
    float *host_matB_halfK = (float *)malloc(total_B_SIZE * sizeof(float)/2);

    // store temp result
    float *host_matD_halfK = (float *)malloc(total_C_SIZE * sizeof(float));

    // final result
    float *host_matD_final_halfK = (float *)malloc(total_C_SIZE * sizeof(float));

    for(int i = 0; i < inst_m*inst_n; i ++){
        host_matD_final_halfK[i] = 0.0f;
        host_matD_halfK[i] = 0.0f;
    }

    for(int i =0; i <2; i++){
        // copy half K to
        // mat a 
        for(int row = 0; row < inst_m; row ++){
            for(int col = 0; col < inst_k/2; col ++){
                host_matA_halfK[col + row*inst_k/2] = matA_in[col + row*inst_k + (inst_k/2) * i];
            }
        }

        for(int row = 0; row < inst_k/2; row ++){
            for(int col = 0; col < inst_n; col ++){
                host_matB_halfK[col + row*inst_n] = matB_in[col + (row + i * inst_k/2)*inst_n ];
            }
        }
        // copy data from host to device
        gpuErrchk(hipMemcpy(dev_matA_halfK, host_matA_halfK, total_A_SIZE * sizeof(float) / 2, hipMemcpyHostToDevice));
    
        gpuErrchk(hipMemcpy(dev_matB_halfK, host_matB_halfK, total_B_SIZE * sizeof(float) / 2, hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(dev_matC_halfK, matC_in, total_C_SIZE * sizeof(float), hipMemcpyHostToDevice));

        gemm_m16n8k4_kernel<<<BLOCKS_NUM, nwarps*warp_size>>>(dev_matA_halfK,dev_matB_halfK,dev_matC_halfK,dev_matD_halfK);
        gpuErrchk(hipPeekAtLastError());
    
        gpuErrchk(hipMemcpy(host_matD_halfK, dev_matD_halfK, total_C_SIZE * sizeof(float), hipMemcpyDeviceToHost));
        
        hipDeviceSynchronize();
        for(int n = 0; n < inst_m*inst_n; n ++){
            host_matD_final_halfK[n] += host_matD_halfK[n];
        }

    }

    copy_mat(matD_out,host_matD_final_halfK,inst_m*inst_n);
    

    hipFree(dev_matA_halfK);
    hipFree(dev_matB_halfK);
    hipFree(dev_matC_halfK);
    hipFree(dev_matD_halfK);

    free(host_matA_halfK);
    free(host_matB_halfK);
    free(host_matD_halfK);
    free(host_matD_final_halfK);

}


std::vector<double> tf32_chain_matmul(int n, random_generator & random_gen){
    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;

    // int BLOCKS_NUM = 1;
    int nwarps = 1;
    //int warp_size = 32;

    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    int num_chain = n;


    float *host_matA = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC = (float *)malloc(total_C_SIZE * sizeof(float));
    float *host_matD = (float *)malloc(total_C_SIZE * sizeof(float));



    // initialize A, row-major
    float *host_matA_cpu = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB_cpu = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC_cpu = (float *)malloc(total_C_SIZE * sizeof(float));


    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_k; c ++){
            //float rnd = (float)(r*inst_k+c);
            float rnd = (float)random_gen();
            #if Init_With_TF32 == 1
            uint32_t* tmp = reinterpret_cast<uint32_t*>(&rnd);
            *tmp = (*tmp  & ~0x1fff);
            #endif
            host_matA_cpu[r*inst_k+c] = rnd;
            host_matA[r*inst_k+c] = (float)rnd;
        }
    }

    for(int r = 0; r < inst_k; r ++){
        for(int c = 0; c < inst_n; c ++){
            float rnd = (float)random_gen();
            #if Init_With_TF32 == 1
            uint32_t* tmp = reinterpret_cast<uint32_t*>(&rnd);
            *tmp = (*tmp  & ~0x1fff);
            #endif
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            host_matB_cpu[r*inst_n+c] = rnd;
            host_matB[r*inst_n+c] = (float)rnd;
        }
    }
    // initialize C, row-major
    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_n; c ++){
            float rnd = 0.0;  //(float)random_gen(gen);
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            //host_matB_cpu[r*inst_n+c] = rnd;
            host_matC_cpu[r*inst_n+c] = (float)rnd;
            host_matC[r*inst_n+c] = float(rnd); //0.0 ;//  (op_CD)random_gen(gen);
        }
    }


    float *cpu_res_baseline = (float *)malloc(total_C_SIZE * sizeof(float));
    float *gpu_m16n8k8 = (float *)malloc(total_C_SIZE * sizeof(float));
    float *gpu_m16n8k4 = (float *)malloc(total_C_SIZE * sizeof(float));
    // host computation

    for(int chain_id =0; chain_id < num_chain; chain_id ++){

        gemm_mnk_cpu(host_matA_cpu,host_matB_cpu,host_matC_cpu,cpu_res_baseline,inst_m,inst_n,inst_k);

        //gpu 
        // gpu tf32 m16b8k8
        gpu_tf32_m16n8k8(host_matA,host_matB,host_matC,gpu_m16n8k8); //(float* matA_in, float * matB_in, float* matC_in, float* matD_out)
        // gpu tf32 m16b8k4
        //gpu_tf32_m16n8k4(host_matA,host_matB,host_matC,gpu_m16n8k4); // (float* matA_in, float * matB_in, float* matC_in, float* matD_out)
        // destination, source
        //copy_mat(host_matA,gpu_m16n8k4); 
        copy_mat(host_matA,gpu_m16n8k8,inst_m*inst_n);
        copy_mat(host_matA_cpu,cpu_res_baseline,inst_m*inst_n);
        // generate new matB and matC

        for(int r = 0; r < inst_k; r ++){
            for(int c = 0; c < inst_n; c ++){
                float rnd = (float)random_gen();
                #if Init_With_TF32 == 1
                uint32_t* tmp = reinterpret_cast<uint32_t*>(&rnd);
                *tmp = (*tmp  & ~0x1fff);
                #endif
                //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
                host_matB_cpu[r*inst_n+c] = rnd;
                host_matB[r*inst_n+c] = (float)rnd;
            }
        }
        // initialize C, row-major
        for(int r = 0; r < inst_m; r ++){
            for(int c = 0; c < inst_n; c ++){
                float rnd = 0.0;  //(float)random_gen(gen);
                //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
                //host_matB_cpu[r*inst_n+c] = rnd;
                host_matC_cpu[r*inst_n+c] = (float)rnd;
                host_matC[r*inst_n+c] = float(rnd); //0.0 ;//  (op_CD)random_gen(gen);
            }
        }

    }

    double l2_relative_err = 0.0;
    compute_diff_l2_norm(cpu_res_baseline,gpu_m16n8k8,inst_m,inst_n,l2_relative_err);


    // double l2_relative_err_halfK = 0.0;
    // compute_diff_l2_norm(cpu_res_baseline,host_matD_final_halfK,inst_m,inst_n,l2_relative_err_halfK);

    std::vector<double> res{l2_relative_err};

    return res;


}




int main(){
    std::cout<<"***********************************"<<std::endl;
    std::cout << "mma tf32 numeric errors with chain matmul"  << std::endl;
    
    std::cout<<"***********************************"<<std::endl;
    if(Init_With_TF32 == 1){
        std::cout<<"Initialization with tf32"<<std::endl;
    }else{
        std::cout<<"Initialization with fp32"<<std::endl;
    }
    
    for(int num_chain = 1; num_chain <= CHAINs; num_chain++){
        double l2_relative_err = 0.0;
        //double l2_relative_err_halfK = 0.0;
        random_generator random_gen(SEED);
        for(int i=0;i < ROUNDS; i ++){
            std::vector<double> errors = tf32_chain_matmul(num_chain,random_gen);
            l2_relative_err += errors[0];
            // l2_relative_err_halfK += errors[1];
        }

        std::cout<<"length of chain = " << num_chain <<", mma.m16n8k8.tf32 l2 relative error :"<< l2_relative_err/ROUNDS << std::endl;
    }

}



/************ chain matmul pseudocode  **************/

// // Define initialization data type
// typedef init_type bf16; // bf16 or fp32
// // Define the chain length
// const int n_chains = N; // e.g. N= 50
// // Initialization for A and B
// initialize_random<init_type>(A_tmp);
// initialize_random<init_type>(B_tmp);
// // Copy to CPU
// Copy_CPU<fp32>(A_CPU,A_tmp);
// Copy_CPU<fp32>(B_CPU,B_tmp);
// // Copy to GPU
// Copy_GPU<bf16>(A_TC,A_tmp);
// Copy_GPU<bf16>(B_TC,B_tmp);
// // Start chain matmul
// for(int i=0; i<n_chains;i++){
//     D_CPU_fp32 = A_CPU*B_CPU ; // run on CPU
//     D_TC = A_TC*B_TC; // run on Tensor Cores
//     // Assign the current result to next Matrix A
//     Copy_CPU<fp32>(A_CPU,D_CPU_fp32);
//     Copy_GPU<bf16>(A_TC,D_TC);
//     // Random generate new Matrix B for next round computation
//     initialize_random<init_type>(B_tmp);
//     Copy_CPU<fp32>(B_CPU,B_tmp);
//     Copy_GPU<bf16>(B_TC,B_tmp);
// }
// // compute the l2_relative error
// l2_err_TC_CPUfp32 = l2_relative_err(D_CPU_fp32,D_TC);


/************ chain matmul pseudocode **************/