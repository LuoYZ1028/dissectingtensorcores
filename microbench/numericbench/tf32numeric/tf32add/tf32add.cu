#include "hip/hip_runtime.h"
// one m16n8k8_tf32 = two m16n8k4 
// see the errors for a same m16n8k8
// chain matrix multiplication
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_fp16.h>
#include <random>
#include "../../../hw_def/hw_def.h"
#include "../../cpu_base.h"
#include <cstring>


// typedef float op_AB; 
// typedef float op_CD; 

// typedef float init_type;
// #define Init_With_TF32 1

typedef float init_type;
#define Init_With_TF32 0

#ifndef MEAN
#define MEAN (0.0)
#endif

#ifndef STDDEV
#define STDDEV (1.0)
#endif

#ifndef ITERS
#define ITERS  (1024 )
#endif

#define ROUNDS  (1000 )


#define SEED 123456

struct random_generator{
    // int seed_;
    // float mean_;
    // float stddev_;
    std::mt19937 gen;
    std::normal_distribution<float> random_gen{MEAN,STDDEV};
    //std::uniform_real_distribution<> random_gen{-1.0,1.0};
    random_generator(int seed_){
        gen.seed(seed_);
    }

    float operator()(){
        return random_gen(gen);
    };
};

// const int inst_m = 16;
const int inst_n = 8;
// const int inst_k = 8;

__forceinline__ __device__ unsigned lane_id_()
{
    unsigned ret; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}





__global__ void gemm_m16n8k8_kernel(float* MatA,float* MatB,float* MatC, float* MatD ){
    // constexpr const int inst_m = 16;
    // constexpr const int inst_n = 8;
    constexpr const int inst_k = 8;
    uint32_t lane_id =  lane_id_();
    // four threads per group, group id
    uint32_t group_id = lane_id >>2;
    uint32_t tid_in_group = lane_id % 4;
    // m16 n8 k8
    uint32_t frag_A[4]; // 16 * 16  / 32 = 8 * bf16
    uint32_t frag_B[2]; // 8 * 16  / 32
    float frag_D[4]; // float , 16*8 /32 = 4*float
    // load operand fragA
    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_a = 0;
        uint32_t col_a = 0;
        if( i==0 || i ==2 ){
            row_a = group_id;
        }else{
            row_a = group_id + 8;
        }
        if(i == 0 || i==1){// i ==0 || i ==2
            col_a = tid_in_group;
        }else{
            col_a = tid_in_group + 4;
        }
        asm("cvt.rna.tf32.f32  %0, %1;\n" : "=r"(frag_A[i]) : "f"(MatA[inst_k*row_a + col_a]));
    }

    #pragma unroll
    for(int i =0; i < 2; i++){
        uint32_t row_b =  0 ; //(i==0)?tid_in_group:(tid_in_group+4);//  (tid_in_group * 2) + (i);
        if(i == 0){
            row_b = tid_in_group;
        }else{
            row_b = tid_in_group + 4;
        }
        uint32_t col_b = group_id;
        // row-major B
        asm("cvt.rna.tf32.f32  %0, %1;\n" : "=r"(frag_B[i]) : "f"(MatB[row_b*inst_n + col_b]));
        //frag_B[i] = (MatB[row_b*inst_n + col_b]);
    }

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_c = 0;
        if( i < 2 ){
            row_c = group_id;
        }else{
            row_c = group_id + 8;
        }
        uint32_t col_c = (tid_in_group * 2) + (i & 0x1);
        // row-major
        frag_D[i] = MatC[inst_n*row_c + col_c];
    }

    uint32_t const *A = reinterpret_cast<uint32_t const *>(&frag_A[0]);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&frag_B[0]);//?
    float *C = reinterpret_cast<float *>(&frag_D[0]);
    float *D = C;  // D = A*B + D.

    asm volatile(
        "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 {%0,%1,%2,%3}, {%4,%5, %6, %7}, {%8,%9}, {%10,%11,%12,%13};\n"
        : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]), 
          "r"(B[0]), "r"(B[1]), 
          "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3])
    );
    
    __syncwarp();

    #pragma unroll
    for(int i =0; i < 4; i++){
        uint32_t row_d = 0;
        if( i < 2 ){
            row_d = group_id;
        }else{
            row_d = group_id + 8;
        }
        uint32_t col_d = (tid_in_group * 2) + (i & 0x1);
        // row-major
        MatD[inst_n*row_d + col_d] = frag_D[i];
    }

}

template<typename typeDest,typename typeSrc >
void copy_mat(typeDest* dest, typeSrc* source, int num_eles){
    for(int i=0;i<num_eles;i++){
        dest[i] = typeDest(source[i]);
    }
}



void gpu_tf32_m16n8k8(float* matA_in, float * matB_in, float* matC_in, float* matD_out){
    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;

    int BLOCKS_NUM = 1;
    int nwarps = 1;
    int warp_size = 32;
    
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    //*********** m16n8k8 device mem allocation***//
    float *dev_matA;
    float *dev_matB;
    float *dev_matC;

    float *dev_matD;
    // allocate device global memory
    // D = A*B + C
    hipMalloc(&dev_matA, total_A_SIZE * sizeof(float));
    hipMalloc(&dev_matB, total_B_SIZE * sizeof(float));
    hipMalloc(&dev_matC, total_C_SIZE * sizeof(float));
    hipMalloc(&dev_matD, total_C_SIZE * sizeof(float));

    /***************************************** m16n8k8 *******************************************/
    // copy data from host to device
    gpuErrchk(hipMemcpy(dev_matA, matA_in, total_A_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(dev_matB, matB_in, total_B_SIZE * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_matC, matC_in, total_C_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gemm_m16n8k8_kernel<<<BLOCKS_NUM, nwarps*warp_size>>>(dev_matA,dev_matB,dev_matC,dev_matD);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(matD_out, dev_matD, total_C_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    hipFree(dev_matA);
    hipFree(dev_matB);
    hipFree(dev_matC);
    hipFree(dev_matD);
}




std::vector<double> tf32_addtion_innerProduct(random_generator & random_gen){
    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;

    // int BLOCKS_NUM = 1;
    int nwarps = 1;
    //int warp_size = 32;
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    // int num_chain = n;
    float *host_matA = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC = (float *)malloc(total_C_SIZE * sizeof(float));
    float *host_matD = (float *)malloc(total_C_SIZE * sizeof(float));


    // initialize A, row-major
    float *host_matA_cpu = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB_cpu = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC_cpu = (float *)malloc(total_C_SIZE * sizeof(float));


    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_k; c ++){
            //float rnd = (float)(r*inst_k+c);


            host_matA_cpu[r*inst_k+c] = 0.0;
            host_matA[r*inst_k+c] = (float)0.0;
        }
    }


    float rnd = (float)random_gen();

    // std::cout<<rnd<<std::endl;

    #if Init_With_TF32 == 1
    uint32_t* tmp = reinterpret_cast<uint32_t*>(&rnd);
    *tmp = (*tmp  & ~0x1fff);
    #endif
    host_matA_cpu[0] = rnd;
    host_matA[0] = rnd;


    rnd = (float)random_gen();
    #if Init_With_TF32 == 1
    uint32_t* tmp2 = reinterpret_cast<uint32_t*>(&rnd);
    *tmp2 = (*tmp2  & ~0x1fff);
    #endif

    host_matA_cpu[1] = rnd;
    host_matA[1] = rnd;


    for(int r = 0; r < inst_k; r ++){
        for(int c = 0; c < inst_n; c ++){
            float rnd = 0.0;
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
   
            host_matB_cpu[r*inst_n+c] = rnd;
            host_matB[r*inst_n+c] = (float)rnd;
        }
    }





    
    host_matB_cpu[0] = 1.0;
    host_matB[0] = 1.0;
    
    host_matB[inst_n] = 1.0;
    host_matB_cpu[inst_n] = 1.0;

    // initialize C, row-major
    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_n; c ++){
            float rnd = 0.0;  //(float)random_gen(gen);
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            //host_matB_cpu[r*inst_n+c] = rnd;
            host_matC_cpu[r*inst_n+c] = (float)rnd;
            host_matC[r*inst_n+c] = float(rnd); //0.0 ;//  (op_CD)random_gen(gen);
        }
    }

    float *cpu_res_baseline = (float *)malloc(total_C_SIZE * sizeof(float));
    float *gpu_m16n8k8 = (float *)malloc(total_C_SIZE * sizeof(float));
    float *gpu_m16n8k4 = (float *)malloc(total_C_SIZE * sizeof(float));
    // host computation
    gemm_mnk_cpu(host_matA_cpu,host_matB_cpu,host_matC_cpu,cpu_res_baseline,inst_m,inst_n,inst_k);

    //gpu 
    // gpu tf32 m16b8k8
    gpu_tf32_m16n8k8(host_matA,host_matB,host_matC,gpu_m16n8k8); //(float* matA_in, float * matB_in, float* matC_in, float* matD_out)

    double l2_relative_err = 0.0;
    compute_diff_l2_norm(cpu_res_baseline,gpu_m16n8k8,inst_m,inst_n,l2_relative_err);


    double abs_err = 0.0;
    abs_err = std::abs(double(cpu_res_baseline[0]) - double(gpu_m16n8k8[0]));

    std::vector<double> res{l2_relative_err,abs_err};

    return res;
}




std::vector<double> tf32_addtion_accumulation(random_generator & random_gen){
    const int inst_m = 16;
    const int inst_n = 8;
    const int inst_k = 8;

    // int BLOCKS_NUM = 1;
    int nwarps = 1;
    //int warp_size = 32;
    unsigned total_A_SIZE = inst_m*inst_k*nwarps;
    unsigned total_B_SIZE = inst_k*inst_n*nwarps;
    unsigned total_C_SIZE = inst_m*inst_n*nwarps;

    // int num_chain = n;
    float *host_matA = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC = (float *)malloc(total_C_SIZE * sizeof(float));
    float *host_matD = (float *)malloc(total_C_SIZE * sizeof(float));


    // initialize A, row-major
    float *host_matA_cpu = (float *)malloc(total_A_SIZE * sizeof(float));
    float *host_matB_cpu = (float *)malloc(total_B_SIZE * sizeof(float));

    float *host_matC_cpu = (float *)malloc(total_C_SIZE * sizeof(float));


    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_k; c ++){
            //float rnd = (float)(r*inst_k+c);


            host_matA_cpu[r*inst_k+c] = 0.0;
            host_matA[r*inst_k+c] = (float)0.0;
        }
    }

    for(int r = 0; r < inst_k; r ++){
        for(int c = 0; c < inst_n; c ++){
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            host_matB_cpu[r*inst_n+c] = 0.0;
            host_matB[r*inst_n+c] = (float)0.0;
        }
    }

    float rnd = (float)random_gen();
    #if Init_With_TF32 == 1
    uint32_t* tmp = reinterpret_cast<uint32_t*>(&rnd);
    *tmp = (*tmp  & ~0x1fff);
    #endif
    host_matA_cpu[0] = rnd;
    host_matA[0] = rnd;


    host_matB_cpu[0] = 1.0;
    host_matB[0] = 1.0;

    // initialize C, row-major
    for(int r = 0; r < inst_m; r ++){
        for(int c = 0; c < inst_n; c ++){
            
            //float rnd = float(r*inst_n+c);   //(float)random_gen(gen);
            //host_matB_cpu[r*inst_n+c] = rnd;
            host_matC_cpu[r*inst_n+c] = 0.0;
            host_matC[r*inst_n+c] = 0.0; //0.0 ;//  (op_CD)random_gen(gen);
        }
    }



    rnd = (float )random_gen();

    // std::cout << "random val fp32 " << rnd << std::endl;

    #if Init_With_TF32 == 1
    uint32_t* tmpc = reinterpret_cast<uint32_t*>(&rnd);
    *tmpc = (*tmpc  & ~0x1fff);
    #endif

    host_matC_cpu[0] = rnd;
    host_matC[0] = rnd;

    
    // std::cout << "convert to tf32 " << rnd << std::endl;



    float *cpu_res_baseline = (float *)malloc(total_C_SIZE * sizeof(float));
    float *gpu_m16n8k8 = (float *)malloc(total_C_SIZE * sizeof(float));
    float *gpu_m16n8k4 = (float *)malloc(total_C_SIZE * sizeof(float));
    // host computation
    gemm_mnk_cpu(host_matA_cpu,host_matB_cpu,host_matC_cpu,cpu_res_baseline,inst_m,inst_n,inst_k);

    //gpu 
    // gpu tf32 m16b8k8
    gpu_tf32_m16n8k8(host_matA,host_matB,host_matC,gpu_m16n8k8); //(float* matA_in, float * matB_in, float* matC_in, float* matD_out)

    double l2_relative_err = 0.0;
    compute_diff_l2_norm(cpu_res_baseline,gpu_m16n8k8,inst_m,inst_n,l2_relative_err);


    double abs_err = 0.0;
    abs_err = std::abs(double(cpu_res_baseline[0]) - double(gpu_m16n8k8[0]));

    std::vector<double> res{l2_relative_err,abs_err};

    return res;
}


int main(){
    

    // std::cout<<"***********************************"<<std::endl;
    // if(Init_With_TF32 == 1){
    //     std::cout<<"Initialization with tf32"<<std::endl;
    // }else{
    //     std::cout<<"Initialization with fp32"<<std::endl;
    // }
    // double l2_relative_err = 0.0;
    // double abs_err = 0.0;
    // for(int i=0;i < ROUNDS; i ++){
    //     std::vector<double> errors = tf32_mul();
    //     l2_relative_err += errors[0];
    //     abs_err+= errors[1];
    // }

    // std::cout<<"mma tf32 with accum fp32 | multiplication | abs numeric errors w.r.t fp32 on CPU = "<< abs_err/ROUNDS << std::endl;
    

    std::cout<<"***********************************"<<std::endl;
    if(Init_With_TF32 == 1){
        std::cout<<"Initialization with tf32"<<std::endl;
    }else{
        std::cout<<"Initialization with fp32"<<std::endl;
    }

    double l2_relative_err = 0.0;
    double abs_err= 0.0;
    random_generator random_gen(SEED);
    for(int i=0;i < ROUNDS; i ++){
        std::vector<double> errors = tf32_addtion_innerProduct(random_gen);
        l2_relative_err += errors[0];
        abs_err += errors[1];
        // l2_relative_err_halfK += errors[1];
    }
    // std::cout<<"mma tf32 with accum fp32 | addition of Inner Product | l2 numeric errors w.r.t fp32 = "<< l2_relative_err/ROUNDS << std::endl;
    std::cout<<"mma tf32 with accum fp32 | addition of Inner Product | abs numeric errors w.r.t fp32 = "<< abs_err/ROUNDS << std::endl;
    //std::cout<<"mma.m16n8k8.bf16 l2 relative error per FMA :"<< l2_relative_err/(ROUNDS*inst_m*inst_k*inst_n) << std::endl;

    std::cout<<"***********************************"<<std::endl;
    l2_relative_err = 0.0;
    abs_err= 0.0;
    random_generator random_gen2(SEED);
    for(int i=0;i < ROUNDS; i ++){
        std::vector<double> errors = tf32_addtion_accumulation(random_gen2);

        // if(errors[2] == 1){ //l2 is inf
        //     i --;
        //     continue;
        // }
        l2_relative_err += errors[0];
        abs_err += errors[1];
        // l2_relative_err_halfK += errors[1];
    }
    // std::cout<<"mma tf32 with accum fp32 | addition of accumulation | l2 numeric errors w.r.t fp32 ="<< l2_relative_err/ROUNDS << std::endl;
    std::cout<<"mma tf32 with accum fp32 | addition of accumulation | abs numeric errors w.r.t fp32 = "<< abs_err/ROUNDS << std::endl;

}



/************ chain matmul pseudocode  **************/







/************ chain matmul pseudocode **************/